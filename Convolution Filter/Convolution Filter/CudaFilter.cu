#include "hip/hip_runtime.h"
#include "CudaFilter.cuh"

#include "Image.h"
#include "Kernel.h"

void *CudaFilter::copyToGRAM(int size, void *data)
{
	void * c_data;
	hipError_t error;

	// Alocate GPU memory
	error = hipMalloc(&c_data, size);
	if (error != hipSuccess)
	{
		hipFree(c_data);
		return nullptr;
	}

	// Copy Data to GPU memory
	error = hipMemcpy(c_data, data, size, hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		hipFree(c_data);
		return nullptr;
	}
	return c_data;

}

__global__ void c_applyFilter(float *kernelData, int kernelWidth, int kernelHeight, void *imgData, int imageWidth, int imageHeight, void * res)
{
	Pixel_t * imageData = (Pixel_t *)imgData;
	Pixel_t * result = (Pixel_t *)res;

	int block = blockIdx.x;
	int thread = threadIdx.x;
	int x1, y1;
	float red, green, blue;

	for (int j = block; j < imageHeight; j += 128)
	{
		for (int b = thread; b < imageWidth; b += 128)
		{
			red = 0;
			green = 0;
			blue = 0;

			for (int x = 0; x < kernelHeight; x++)
			{
				for (int y = 0; y < kernelWidth; y++)
				{
					x1 = b - kernelWidth / 2 + x;
					y1 = j - kernelHeight / 2 + y;

					if ((x1 < 0) || (x1 >= imageWidth))
					{
						x1 = (imageWidth - x1) % imageWidth;
					}

					if ((y1 < 0) || (y1 >= imageHeight))
					{
						y1 = (imageHeight - y1) % imageHeight;
					}

					red += imageData[y1*imageWidth + x1].red * kernelData[x*kernelWidth + y];
					green += imageData[y1*imageWidth + x1].green * kernelData[x*kernelWidth + y];
					blue += imageData[y1*imageWidth + x1].blue * kernelData[x*kernelWidth + y];
				}
			}

			red = red > 0 ? (red > 255 ? 255 : red) : 0;
			green = green > 0 ? (green > 255 ? 255 : green) : 0;
			blue = blue > 0 ? (blue > 255 ? 255 : blue) : 0;

			result[j*imageWidth + b].red = (unsigned char)red;
			result[j*imageWidth + b].green = (unsigned char)green;
			result[j*imageWidth + b].blue = (unsigned char)blue;
		}
	}
}


Image * CudaFilter::applyFilter()
{
	int imageSize = image->getHeight()*image->getWidth() * sizeof(Pixel_t);
	int kernelSize = kernel->getHeight()*kernel->getWidth() * sizeof(float);

	// Copy data to GPU memory
	void * c_kernel = copyToGRAM(kernelSize, kernel->getData());
	void * c_image = copyToGRAM(imageSize, image->getData());
	void * c_result;

	hipMalloc(&c_result, imageSize);
	hipDeviceSynchronize();

	if (c_kernel == nullptr || c_image == nullptr)
		return nullptr;

	c_applyFilter << < 128, 128 >> > ((float *)c_kernel, kernel->getWidth(), kernel->getHeight(), (Pixel_t *)c_image, image->getWidth(), image->getHeight(), (Pixel_t *)c_result);
	hipDeviceSynchronize();

	Pixel_t *result = new Pixel_t[image->getWidth()*image->getHeight()];
	hipMemcpy(result, c_result, imageSize, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipFree(c_kernel);
	hipFree(c_image);
	hipFree(c_result);

	return new Image(image->getWidth(), image->getHeight(), result);
}
